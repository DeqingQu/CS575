#include "hip/hip_runtime.h"
// Array multiplication: C = A * B:

// System includes
#include <stdio.h>
#include <assert.h>
#include <malloc.h>
#include <math.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"


#ifndef BLOCKSIZE
#define BLOCKSIZE		32		// number of threads per block
#endif

#ifndef NUMTRIALS
#define NUMTRIALS			16*1024	// array size
#endif

#ifndef TOLERANCE
#define TOLERANCE		0.00001f	// tolerance to relative error
#endif

// ranges for the random numbers:
const float XCMIN =	 0.0;
const float XCMAX =	 2.0;
const float YCMIN =	 0.0;
const float YCMAX =	 2.0;
const float RMIN  =	 0.5;
const float RMAX  =	 2.0;

// array multiplication (CUDA Kernel) on the device: C = A * B

__global__  void ArrayMul( float *A, float *B, float *C )
{
	int gid = blockIdx.x*blockDim.x + threadIdx.x;
	C[gid] = A[gid] * B[gid];
}

__global__  void MonteCarlo( float *xcs, float *ycs, float *rs, int *numHits )
{
	//int gid = blockIdx.x*blockDim.x + threadIdx.x;
	//C[gid] = A[gid] * B[gid];
    //numHits[0] = 10000;
        
    __shared__ float prods[BLOCKSIZE];

	unsigned int numItems = blockDim.x;
	unsigned int tnum = threadIdx.x;
	unsigned int wgNum = blockIdx.x;
	unsigned int gid = blockIdx.x*blockDim.x + threadIdx.x;

	prods[tnum] = 1;

	for (int offset = 1; offset < numItems; offset *= 2)
	{
		int mask = 2 * offset - 1;
		__syncthreads();
		if ((tnum & mask) == 0)
		{
			prods[tnum] += prods[tnum + offset];
		}
	}

	__syncthreads();
	if (tnum == 0)
		numHits[wgNum] = prods[0];
}

// helper functions
float Ranf( float low, float high )
{
        float r = (float) rand();               // 0 - RAND_MAX
        float t = r  /  (float) RAND_MAX;       // 0. - 1.

        return   low  +  t * ( high - low );
}

int Ranf( int ilow, int ihigh )
{
        float low = (float)ilow;
        float high = ceil( (float)ihigh );

        return (int) Ranf(low,high);
}

void TimeOfDaySeed( )
{
	struct tm y2k = { 0 };
	y2k.tm_hour = 0;   y2k.tm_min = 0; y2k.tm_sec = 0;
	y2k.tm_year = 100; y2k.tm_mon = 0; y2k.tm_mday = 1;

	time_t  timer;
	time( &timer );
	double seconds = difftime( timer, mktime(&y2k) );
	unsigned int seed = (unsigned int)( 1000.*seconds );    // milliseconds
	srand( seed );
}

// main program:

int
main( int argc, char* argv[ ] )
{
	int dev = findCudaDevice(argc, (const char **)argv);

	// allocate host memory:

//	float * hA = new float [ SIZE ];
//	float * hB = new float [ SIZE ];
//	float * hC = new float [ SIZE ];
    
    
	float * hxcs = new float [ NUMTRIALS ];
	float * hycs = new float [ NUMTRIALS ];
	float * hrs = new float [ NUMTRIALS ];
    int * hnumHits = new int [ NUMTRIALS/BLOCKSIZE ];
    

    // fill the random-value arrays:
    for( int n = 0; n < NUMTRIALS; n++ )
    {
        hxcs[n] = Ranf( XCMIN, XCMAX );
        hycs[n] = Ranf( YCMIN, YCMAX );
        hrs[n] = Ranf(  RMIN,  RMAX );
    }
    
//	for( int i = 0; i < SIZE; i++ )
//	{
//		hA[i] = hB[i] = (float) sqrt(  (float)i  );
//	}

	// allocate device memory:

//	float *dA, *dB, *dC;
    float *dxcs, *dycs, *drs;
    int *dnumHits;

//	dim3 dimsA( SIZE, 1, 1 );
//	dim3 dimsB( SIZE, 1, 1 );
//	dim3 dimsC( SIZE, 1, 1 );
    
	dim3 dimsxcs( NUMTRIALS, 1, 1 );
	dim3 dimsycs( NUMTRIALS, 1, 1 );
	dim3 dimsrs( NUMTRIALS, 1, 1 );
    dim3 dimsnumHits( NUMTRIALS, 1, 1);
    

	hipError_t status;
//	status = hipMalloc( reinterpret_cast<void **>(&dA), SIZE*sizeof(float) );
//		checkCudaErrors( status );
//	status = hipMalloc( reinterpret_cast<void **>(&dB), SIZE*sizeof(float) );
//		checkCudaErrors( status );
//	status = hipMalloc( reinterpret_cast<void **>(&dC), SIZE*sizeof(float) );
//		checkCudaErrors( status );

	status = hipMalloc( reinterpret_cast<void **>(&dxcs), NUMTRIALS*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&dycs), NUMTRIALS*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&drs), NUMTRIALS*sizeof(float) );
		checkCudaErrors( status );
    status = hipMalloc( reinterpret_cast<void **>(&dnumHits), (NUMTRIALS/BLOCKSIZE)*sizeof(int) );
		checkCudaErrors( status );


	// copy host memory to the device:

//	status = hipMemcpy( dA, hA, SIZE*sizeof(float), hipMemcpyHostToDevice );
//		checkCudaErrors( status );
//	status = hipMemcpy( dB, hB, SIZE*sizeof(float), hipMemcpyHostToDevice );
//		checkCudaErrors( status );

	status = hipMemcpy( dxcs, hxcs, NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );
	status = hipMemcpy( dycs, hycs, NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );
    status = hipMemcpy( drs, hrs, NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );

	// setup the execution parameters:

	dim3 threads(BLOCKSIZE, 1, 1 );
	dim3 grid( NUMTRIALS / threads.x, 1, 1 );

	// Create and start timer

	hipDeviceSynchronize( );

	// allocate CUDA events that we'll use for timing:

	hipEvent_t start, stop;
	status = hipEventCreate( &start );
		checkCudaErrors( status );
	status = hipEventCreate( &stop );
		checkCudaErrors( status );

	// record the start event:

	status = hipEventRecord( start, NULL );
		checkCudaErrors( status );

	// execute the kernel:

//	ArrayMul<<< grid, threads >>>( dA, dB, dC );
    MonteCarlo<<< grid, threads >>>( dxcs, dycs, drs, dnumHits );

	// record the stop event:

	status = hipEventRecord( stop, NULL );
		checkCudaErrors( status );

	// wait for the stop event to complete:

	status = hipEventSynchronize( stop );
		checkCudaErrors( status );

	float msecTotal = 0.0f;
	status = hipEventElapsedTime( &msecTotal, start, stop );
		checkCudaErrors( status );

	// compute and print the performance

	double secondsTotal = 0.001 * (double)msecTotal;
	double trialsPerSecond = (float)NUMTRIALS / secondsTotal;
	double megaTrialsPerSecond = trialsPerSecond / 1000000.;
	fprintf( stderr, "Size = %10d, MegaTrials/Second = %10.2lf\n", NUMTRIALS, megaTrialsPerSecond );

	// copy result from the device to the host:

	status = hipMemcpy( hnumHits, dnumHits, (NUMTRIALS/BLOCKSIZE)*sizeof(int), hipMemcpyDeviceToHost );
		checkCudaErrors( status );
    
    int sum = 0.;
	for(int i = 0; i < NUMTRIALS/BLOCKSIZE; i++ )
	{
		sum += hnumHits[i];
	}
    float prob = (float)(sum) / (float)(NUMTRIALS);
	fprintf( stderr, "\nProbability=%8.4lf\n", prob );
    fprintf( stderr, "\NumHit=%10d\n", sum );
    
	// clean up memory:
//	delete [ ] hA;
//	delete [ ] hB;
//	delete [ ] hC;
    delete [ ] hxcs;
    delete [ ] hycs;
    delete [ ] hrs;
    delete [ ] hnumHits;

//	status = hipFree( dA );
//		checkCudaErrors( status );
//	status = hipFree( dB );
//		checkCudaErrors( status );
//	status = hipFree( dC );
//		checkCudaErrors( status );

    status = hipFree( dxcs );
		checkCudaErrors( status );
	status = hipFree( dycs );
		checkCudaErrors( status );
	status = hipFree( drs );
		checkCudaErrors( status );
    status = hipFree( dnumHits );
		checkCudaErrors( status );



	return 0;
}
